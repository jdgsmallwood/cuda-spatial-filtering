#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <algorithm>

__global__ void convert_int8_to_half_kernel(const int8_t *d_input, __half *d_output, const int n) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    while (idx < n) {
        // Promote int8_t to int before conversion
        int val = static_cast<int>(d_input[idx]);
        d_output[idx] = __int2half_rn(val);
        idx += stride;
    }
}

__global__ void update_weights_kernel(const __half *d_weights, __half *d_weights_output, const int num_beams, const int num_receivers, const int num_channels) {
    
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    const int n = num_beams * num_receivers;

    while (idx < n) {
        d_weights_output[idx] = d_weights[idx];
        idx += stride;
    }

}


__global__ void convert_int_to_float_kernel(const int *d_input, float *d_output, const int n) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    while (idx < n) {
        d_output[idx] = __int2float_rn(d_input[idx]);
        idx += stride;
    }   

}

void convert_int8_to_half(const int8_t *d_input, __half *d_output, const int n, hipStream_t &stream) {
    
    const int num_blocks = std::min(4, n / 1024 + 1);

    convert_int8_to_half_kernel<<<num_blocks, 1024, 0, stream>>>(d_input, d_output, n);

}


void convert_int_to_float(const int *d_input, float *d_output, const int n, hipStream_t &stream) {
    
    const int num_blocks = std::min(4, n / 1024 + 1);

    convert_int_to_float_kernel<<<num_blocks, 1024, 0, stream>>>(d_input, d_output, n);


}


void update_weights(const __half *d_weights, __half *d_weights_output, const int num_beams, const int num_receivers,const int num_channels, const float *d_eigenvalues, const float *d_eigenvectors, hipStream_t &stream) {

    const int n = num_beams * num_receivers * num_channels;
    const int num_blocks = std::min(4, n / 1024 + 1);

    update_weights_kernel<<<num_blocks, 1024, 0, stream>>>(d_weights, d_weights_output, num_beams, num_receivers, num_channels);
}


