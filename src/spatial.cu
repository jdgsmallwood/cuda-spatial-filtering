#include "hip/hip_runtime.h"
#include "spatial/spatial.hpp"
#include <hip/hip_runtime.h>

__global__ void increment_kernel(int* data, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        data[idx]++;
    }
}

void incrementArray(int* data, int size) {
    int* d_data = nullptr;
    hipMalloc(&d_data, size * sizeof(int));
    hipMemcpy(d_data, data, size * sizeof(int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    increment_kernel<<<blocks, threads>>>(d_data, size);
    hipDeviceSynchronize();

    hipMemcpy(data, d_data, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_data);
}