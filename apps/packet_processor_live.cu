#include "hip/hip_runtime.h"
#include "spatial/logging.hpp"
#include "spatial/packet_formats.hpp"
#include "spatial/pipeline.hpp"
#include "spatial/spatial.hpp"
#include <algorithm>
#include <arpa/inet.h>
#include <atomic>
#include <complex>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <highfive/highfive.hpp>
#include <iostream>
#include <spdlog/sinks/basic_file_sink.h>
#include <stdexcept>
#include <string>
#include <sys/socket.h>
#include <sys/time.h>
#include <thread>
#include <unistd.h>

// Processor thread - continuously processes packets

void print_startup_info() {

  // Startup debug info
  LOG_INFO("NR_CHANNELS: {}", NR_CHANNELS_DEF);
  LOG_INFO("NUM_FRAMES_PER_ITERATION: {}", NUM_FRAMES_PER_ITERATION);
  LOG_INFO("NR_TOTAL_FRAMES_PER_CHANNEL: {}", NR_TOTAL_FRAMES_PER_CHANNEL);
  LOG_INFO("NR_FPGA_SOURCES: {}", NR_FPGA_SOURCES);
  LOG_INFO("NR_RECEIVERS: {}", NR_RECEIVERS_DEF);
  LOG_INFO("NR_RECEIVERS_PER_PACKET: {}", NR_RECEIVERS_DEF_PER_PACKET);
  LOG_INFO("NR_PACKETS_FOR_CORRELATION: {}", NR_PACKETS_FOR_CORRELATION);
  LOG_INFO("NR_INPUT_BUFFERS: {}", NR_INPUT_BUFFERS);
  LOG_INFO("PacketDataStructure size is {}", sizeof(PacketDataStructure));
  LOG_INFO("PacketScaleStructure size is {}", sizeof(PacketScaleStructure));
}

int main() {
  auto app_logger = spdlog::basic_logger_mt("packet_processor_live_logger",
                                            "app.log", /*truncate*/ true);
  app_logger->set_level(spdlog::level::debug);
  app_logger->set_pattern("[%Y-%m-%d %H:%M:%S] [%l] %v");

  // Provide it to the library
  spatial::Logger::set(app_logger);
  print_startup_info();

  ProcessorState<LambdaPacketStructure> state;

  int num_buffers = 2;
  constexpr int num_lambda_channels = 8;
  constexpr int nr_lambda_polarizations = 2;
  constexpr int nr_lambda_receivers = 10;
  constexpr int nr_lambda_beams = 8;
  BeamWeights<num_lambda_channels, nr_lambda_receivers, nr_lambda_polarizations,
              nr_lambda_beams>
      h_weights;

    for (auto i = 0; i < num_lambda_channels; ++i) {
        for (auto j =0; j <nr_lambda_receivers; ++j) {
            for (auto k = 0; k < nr_lambda_beams; ++ k) {
                for (auto l = 0; l < nr_lambda_polarizations; ++l) {
                    h_weights.weights[i][l][k][j] = 1 / nr_lambda_receivers;
                }
            }
        }
    }

  



  LambdaGPUPipeline<
      /* data input bits */ 8,
      /* channels */ num_lambda_channels,
      /* time steps per packet */ 64,
      /* packets for correlation */ 16,
      /*nr receivers */ 10,
      /* padded receivers (round up to * of 32) */ 32,
      /* nr polarizations */ 2,
      /* nr beams */ 8,
      /* receivers per block */ 32>
      pipeline(num_buffers, &h_weights);

  state.set_pipeline(&pipeline);
  pipeline.set_state(&state);
  int port = 12345;
  KernelSocketPacketCapture socket_capture(port, BUFFER_SIZE);
  LOG_INFO("Ring buffer size: {} packets\n", RING_BUFFER_SIZE);
  LOG_INFO("Starting threads....");
  std::thread receiver(
      [&socket_capture, &state]() { socket_capture.get_packets(state); });

  std::thread processor([&state]() { state.process_packets(); });

  std::cout << "Setup completed. Ready to receive!" << std::endl;
  // Print statistics periodically
  while (state.running) {
    sleep(5);
    // This is nice to see outside of log files.
    std::cout << "Stats: Received=" << state.packets_received
              << ", Processed=" << state.packets_processed << std::endl;
  }

  // Cleanup
  LOG_INFO("\nShutting down...\n");
  state.running = 0;
  receiver.join();
  processor.join();
  app_logger->flush();
  return 0;
}
